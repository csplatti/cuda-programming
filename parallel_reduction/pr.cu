
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <chrono>

using namespace std;

__global__ void parallelSum(int *nums, int N) {
    int i = threadIdx.x;

    // printf("Thread id: %d\n", i);

    float indexJump = N / 2;
    int roundedJump = ceilf(indexJump);


    if (2 * i <= N) {
        nums[i] += nums[i+roundedJump];
        // printf("i: %d; 2i: %d\n", nums[i], nums[2*i]);
    }

    float nextSize = N / 2;
    int nextN = ceilf(nextSize);
    if (nextN > 1 && i == 0) {
        // printf("Nextn: %d\n", nextN);
        parallelSum<<<1, nextN>>>(nums, nextN); // this runs in each thread: that is bad
    }
}

// REQUIRES: N == length of nums
void serialSum(int nums[], int numsSize, int *serialOut) {
    int sum = 0;
    for (int i = 0; i < numsSize; i++) {
        sum += nums[i];
    }

    *serialOut = sum;
}

int main() {
    int nums[20];
    int numsSize = sizeof(nums) / sizeof(int);

    for (int i = 0; i < numsSize; i++) {
        nums[i] = i+1;
        // cout << nums[i] << endl;
    }

    int serialSumResult = 0;

    auto startSerial = chrono::high_resolution_clock::now();
    serialSum(nums, numsSize, &serialSumResult);
    auto endSerial = chrono::high_resolution_clock::now();

    auto serialTime = chrono::duration_cast<chrono::milliseconds>(endSerial-startSerial);

    cout << "Serial: " << serialSumResult << " Runtime: " << serialTime.count() << "ms" << endl;

    int parallelSumResult = 0;

    int *pn = 0;

    hipMalloc(&pn, sizeof(nums));
    hipMemcpy(pn, nums, sizeof(nums), hipMemcpyHostToDevice);

    int numThreads = ceil(numsSize / 2);

    auto startParallel = chrono::high_resolution_clock::now();
    parallelSum<<<1, numThreads>>>(pn, numsSize);
    auto endParallel = chrono::high_resolution_clock::now();

    auto parallelTime = chrono::duration_cast<chrono::milliseconds>(endParallel - startParallel);

    hipMemcpy(nums, pn, sizeof(nums), hipMemcpyDeviceToHost);
    parallelSumResult = nums[0];

    cout << "Parallel: " << parallelSumResult << " Runtime: " << parallelTime.count() << "ms" << endl;
}