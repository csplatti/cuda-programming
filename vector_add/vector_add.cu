
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

__global__ void addVectors(int *a, int *b, int *c, int *n) {
    int i = threadIdx.x;
    c[i] = a[i] + b[i];

    n[i] = sizeof(c) / sizeof(int); 
    // this is not what you are expecting it to be because you cannot find the length of an array from a pointer!
}

int main() {
    int a[] = {1, 2, 3, 4, 5};
    int b[] = {5, 4, 3, 2, 1};
    int c[] = {0, 0, 0, 0, 0};
    int n[] = {0, 0, 0, 0, 0};

    int *pa = 0;
    int *pb = 0;
    int *pc = 0;
    int *pn = 0;

    hipMalloc(&pa, sizeof(a));
    hipMalloc(&pb, sizeof(b));
    hipMalloc(&pc, sizeof(c));
    hipMalloc(&pn, sizeof(n));

    hipMemcpy(pa, a, sizeof(a), hipMemcpyHostToDevice);
    hipMemcpy(pb, b, sizeof(b), hipMemcpyHostToDevice);
    hipMemcpy(pc, c, sizeof(c), hipMemcpyHostToDevice);
    hipMemcpy(pn, n, sizeof(n), hipMemcpyHostToDevice);

    addVectors<<<1, sizeof(c)>>>(pa, pb, pc, pn);

    hipMemcpy(c, pc, sizeof(c), hipMemcpyDeviceToHost);
    hipMemcpy(n, pn, sizeof(n), hipMemcpyDeviceToHost);

    for (int i = 0; i < sizeof(c) / sizeof(int); i++) {
        cout << c[i];
    }
    cout << endl;

    for (int i = 0; i < sizeof(n) / sizeof(int); i++) {
        cout << n[i];
    }
    cout << endl;
}